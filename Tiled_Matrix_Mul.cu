//Tiling done, but concept still not that visible

#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

// Static shmem calculation for convenience (Int 16x16 matrix)
#define SHMEM_SIZE 16 * 16 * 4

__global__ void tiledMatrixMul(int *a, int *b, int *c, int n, int tile_size) {
	// Two statically-sized pieces of shared memory
	__shared__ int A[SHMEM_SIZE];
	__shared__ int B[SHMEM_SIZE];

	// Shorten these parameters for clean re-use
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Calculate global row and column positions for this thread
	int row = by * tile_size + ty;
	int col = bx * tile_size + tx;

	// Intermediate sum for element being written
	int temp_val = 0;

	// Sweep tiles over entire matrix
	for (int i = 0; i < (n / tile_size); i++) {
		/*
			Every thread in a threadblock loads one element into shared memory
			The element location in shared memory corresponds to the thread's
			position in the threadblock (e.g. thread [0, 0] loads for 
			A[0 * tile_size + 0], and B[0 * tile_size + 0].)
			Explanation of indexing parameters
			For A:
				        row*n: Indexes the global row for this thread (loop-invariant)
				  i*tile_size: Indexes the new set of columns each iteration
				           tx: Indexes the column within that set
			for B:
				i*tile_size*n: Indexes the next set of rows each iteration
				         ty*n: Indexes the row within that set
						  col: Indexes the global column (loop-invariant)
		*/
		A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
		B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col];

		// Ensure all threads have loaded their data before proceeding
		__syncthreads();

		// Calculate all temp values for this tile
		for (int j = 0; j < tile_size; j++) {
			temp_val += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
		}

		// Ensure some threads don't progress and stomp current shared memory values
		__syncthreads();
	}
	c[(row * n) + col] = temp_val;
}

void check_answer(int *a, int *b, int *c, int n) {
	int tmp;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			tmp = 0;
			for (int k = 0; k < n; k++) {
				 tmp += a[i * n + k] * b[k * n + j];
			}
            assert(tmp == c[i * n + j]);
		}
	}
}

void init_matrix(int *a, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 10;
		}
	}
}

int main() {
	// Problem size = 1024 x 1024 matrix
	int n = 1 << 10;

	// Matrix size (in bytes)
	size_t bytes = n * n * sizeof(int);

	// Host matrix pointers
	int *h_a, *h_b, *h_c;

	// Device matrix pointers
	int *d_a, *d_b, *d_c;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Allocate device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize matrices
	init_matrix(h_a, n);
	init_matrix(h_b, n);

	// Copy matrices to the device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block (in both x and y dimensions)
	int BLOCK_SIZE = 16;

	// Blocks in each dimension
	int GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

	// Use dim3 objects for 2-D grids and threadblocks
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	tiledMatrixMul <<<grid, threads>>> (d_a, d_b, d_c, n, BLOCK_SIZE);

	// Copy result back from device
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);	

	// Verify the result
	check_answer(h_a, h_b, h_c, n);

	// Free host memory
	free(h_a);
	free(h_b);
	free(h_c);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
